#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <sstream>
#include "matrix_multiplication.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

// Use a 2D grid: each thread computes a value of the output matrix,
// i.e. the dot-product between x[i, :] and y[:, j];
__global__ void gpu_matrix_multiplication_0(double* x, double* y, double* z, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        double sum = 0;
        for (int k = 0; k < N; k++) {                
            sum += x[i * N + k] * y[k * N + j];
        }
        z[i * N + j] = sum;
    }
}

// Same as the first implementation, but with grid-stride loops;
__global__ void gpu_matrix_multiplication_1(double* x, double* y, double* z, int N) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        for(int j = blockIdx.y * blockDim.y + threadIdx.y; j < N; j += blockDim.y * gridDim.y) {
            double sum = 0;
            for (int k = 0; k < N; k++) {                
                sum += x[i * N + k] * y[k * N + j];
            }
            z[i * N + j] = sum;
        }
    }
}

// #define BLOCK_DIM 16
// __global__ void gpu_matrix_multiplication_2(double* x, double* y, double* z, int N) {

//     int tile_size = BLOCK_DIM;
//     int z_block_i = blockIdx.x;
//     int z_block_j = blockIdx.y;
//     // Coordinate of the Z matrix element computed by this specific thread, with respect to the current tile;
//     int z_i = threadIdx.x;
//     int z_j = threadIdx.y;
//     // Coordinate of the Z matrix element computed by this specific thread, with respect to the overall Z matrix (not counting host-level data partitioning);
//     int i = z_block_i * blockDim.x + threadIdx.x;
//     int j = z_block_j * blockDim.y + threadIdx.y;

//     // Value of the Z matrix block being computed by this specific thread;
//     double z_val_ij = 0;

//     // Loop over the tiles in the same row (for X) and column (for Y) of the desired output tile in Z;
//     for (int curr_block_index = 0; curr_block_index < N / tile_size; curr_block_index++) {
//         // Shared memory used to store the current tiles of X and Y;
//         __shared__ double x_tile[BLOCK_DIM][BLOCK_DIM];
//         __shared__ double y_tile[BLOCK_DIM][BLOCK_DIM];
//         x_tile[z_i][z_j] = x[N * i + curr_block_index * tile_size + z_j];
//         y_tile[z_i][z_j] = y[N * (z_i + curr_block_index * tile_size) + j];
//         __syncthreads();
//         // Multiply the i row and j column of the tile;
//         for (int k = 0; k < tile_size; k++) {   
//             z_val_ij += x_tile[z_i][k] * y_tile[k][z_j];
//         }
//         __syncthreads();
//     }
//     z[i * N + j] = z_val_ij;
// }

// Better implementation, using shared memory to compute square tiles of z;
__global__ void gpu_matrix_multiplication_2(double* x, double* y, double* z, int N) {

    // int tile_size = BLOCK_DIM;
    int tile_size = blockDim.x;

    // In the simplest implementation, each block computes a tile of the Z matrix, 
    // whose coordinates are given by blockIdx.x and blockIdx.y;
    // Here, we allow each block to process more tiles, hence the loops below;
    for(int z_block_i = blockIdx.x; z_block_i < (N + tile_size - 1) / tile_size; z_block_i += gridDim.x) {
        for(int z_block_j = blockIdx.y; z_block_j < (N + tile_size - 1) / tile_size; z_block_j += gridDim.y) {
            // Coordinate of the Z matrix element computed by this specific thread, with respect to the current tile;
            int z_i = threadIdx.x;
            int z_j = threadIdx.y;
            // Coordinate of the Z matrix element computed by this specific thread, with respect to the overall Z matrix (not counting host-level data partitioning);
            int i = z_block_i * blockDim.x + threadIdx.x;
            int j = z_block_j * blockDim.y + threadIdx.y;

            // Value of the Z matrix block being computed by this specific thread;
            double z_val_ij = 0;

            // Loop over the tiles in the same row (for X) and column (for Y) of the desired output tile in Z;
            for (int curr_block_index = 0; curr_block_index < (N + tile_size - 1) / tile_size; curr_block_index++) {
                // Shared memory used to store the current tiles of X and Y;
                extern __shared__ double tiles[];
                double *x_tile = tiles;
                double *y_tile = tiles + tile_size * tile_size;
                // Each thread in the block loads a value into the tile;
                if ((i < N) && (curr_block_index * tile_size + z_j < N)) {
                    x_tile[z_i * tile_size + z_j] = x[N * i + curr_block_index * tile_size + z_j];
                } else {
                    x_tile[z_i * tile_size + z_j] = 0;
                }
                if ((j < N) && (curr_block_index * tile_size + z_i < N)) {
                    y_tile[z_i * tile_size + z_j] = y[N * (curr_block_index * tile_size + z_i) + j];
                } else {
                    y_tile[z_i * tile_size + z_j] = 0;
                }
                // Synchronize threads in the block, ensure the tile has been loaded;
                __syncthreads();

                // Multiply the i row and j column of the tile;
                for (int k = 0; k < tile_size; k++) {   
                    z_val_ij += x_tile[z_i * tile_size + k] * y_tile[k * tile_size + z_j];
                }

                // Synchronize threads in the block, ensure the computation has finished before loading the next tile;
                __syncthreads();
            }
            // Write the output value into Z, keeping into account the offset of the current block;
            if ((i < N) & (j < N)) {
                z[i * N + j] = z_val_ij;
            } 
        }
    }
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void MatrixMultiplication::alloc() {
    // All square N x N matrices, with data stored as row-major;

    // Compute the number of blocks for implementations where the value is a function of the input size.
    // Note that in this case the grid is composed of (B x B) blocks, each with size (block_size x block_size);
    B = (N + block_size - 1) / block_size;
    
    // Allocate CPU data;
    x = (double*) malloc(sizeof(double) * N * N);
    y = (double*) malloc(sizeof(double) * N * N);
    z = (double*) malloc(sizeof(double) * N * N);
   
    // Allocate GPU data;
    err = hipMalloc(&x_d, sizeof(double) * N * N);
    err = hipMalloc(&y_d, sizeof(double) * N * N);
    err = hipMalloc(&z_d, sizeof(double) * N * N);
}

// Initialize data;
void MatrixMultiplication::init() {
    // X and Y contains the same data
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            x[i * N + j] = double(i * N + j) / (N * N);
            y[i * N + j] = double(i * N + j) / (N * N);
        }
    }
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void MatrixMultiplication::reset() {
    // Reset the result;
    memset(z, 0, sizeof(double) * N * N);
    // Reset result on the GPU;
    hipMemset(z_d, 0, sizeof(double) * N * N);
    // Transfer data to the GPU;
    hipMemcpy(x_d, x, sizeof(double) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, sizeof(double) * N * N, hipMemcpyHostToDevice);
}

void MatrixMultiplication::matrix_multiplication_0(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation. In this case, the number of blocks depends on N;
    dim3 block_num_2d(B, B);
    dim3 block_size_2d(block_size, block_size);
    gpu_matrix_multiplication_0<<<block_num_2d, block_size_2d>>>(x_d, y_d, z_d, N);

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (3 * sizeof(double) * N * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(z, z_d, sizeof(double) * N * N, hipMemcpyDeviceToHost);
}

void MatrixMultiplication::matrix_multiplication_1(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation. In this case, the number of blocks is chosen by the user;
    dim3 block_num_2d(num_blocks, num_blocks);
    dim3 block_size_2d(block_size, block_size);
    gpu_matrix_multiplication_1<<<block_num_2d, block_size_2d>>>(x_d, y_d, z_d, N);

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (3 * sizeof(double) * N * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(z, z_d, sizeof(double) * N * N, hipMemcpyDeviceToHost);
}

void MatrixMultiplication::matrix_multiplication_2(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation. In this case, the number of blocks is chosen by the user;
    dim3 block_num_2d(num_blocks, num_blocks);
    dim3 block_size_2d(block_size, block_size);
    gpu_matrix_multiplication_2<<<block_num_2d, block_size_2d, 2 * sizeof(double) * block_size * block_size>>>(x_d, y_d, z_d, N);

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (3 * sizeof(double) * N * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(z, z_d, sizeof(double) * N * N, hipMemcpyDeviceToHost);
}

// In this example, use CUBLAS;
void MatrixMultiplication::matrix_multiplication_3(int iter) {
    auto start_tmp = clock_type::now();
    double alpha = 1;
    double beta = 0;
    // You can pass HIPBLAS_OP_T (instead of HIPBLAS_OP_N) to say that matrices should be transposed;
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, x_d, N, y_d, N, &beta, z_d, N);
    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << double(exec_time) / 1000 << " ms, " << (3 * sizeof(double) * N * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(z, z_d, sizeof(double) * N * N, hipMemcpyDeviceToHost);
}

void MatrixMultiplication::execute(int iter) {
    switch (implementation)
    {
    case 0:
        matrix_multiplication_0(iter);
        break;
    case 1:
        matrix_multiplication_1(iter);
        break;
    case 2:
        matrix_multiplication_2(iter);
        break;
    case 3:
        matrix_multiplication_3(iter);
        break;
    default:
        break;
    }
}

// Compute the sum of values of CPU and GPU results, and use that as checksum.
// Also count the number of different CPU and GPU results.
// The CPU implementation does not really perform a standard matrix multiplication,
// but the memory access pattern is identical to a standard matrix multiplication,
// and execution time should be similar as well;
void MatrixMultiplication::cpu_validation(int iter) {
    auto start_tmp = clock_type::now();
    unsigned int errors = 0;
    double sum_total = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            double sum = 0;
            for (int k = 0; k < N; k++) {
                sum += x[i * N + k] * y[k * N + j];
            }
            double z_res = z[i * N + j];
            sum_total += sum / (N * N);
            gpu_checksum += z_res / (N * N);
            if (std::abs(z_res - sum) > 1e-4) {
                errors += 1;
            }
        }
    }
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << double(exec_time) / 1000 << " ms" << std::endl;
    if (errors > 0) std::cout << "result error! errors=" << errors << "; GPU checksum=" << gpu_checksum << ", CPU checksum=" << sum_total << std::endl; 
}

std::string MatrixMultiplication::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(z[0]);
    } else {
        std::ostringstream out;
        out.precision(3);
        out << "[\n";
        for (int i = 0; i < std::min(5, N); i++) {
            out << "[";
            for (int j = 0; j < std::min(5, N); j++) {
                out << z[i * N + j] << ", ";
            }
            out << "...]\n";
        }
        out << "...]";
        return out.str();
    }
}

void MatrixMultiplication::clean() {
    free(x);
    free(y);
    free(z);
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
    hipblasDestroy(handle);
}
