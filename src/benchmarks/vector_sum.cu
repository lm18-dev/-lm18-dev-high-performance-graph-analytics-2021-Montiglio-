#include "hip/hip_runtime.h"
// Copyright (c) 2020, 2021, NECSTLab, Politecnico di Milano. All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NECSTLab nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//  * Neither the name of Politecnico di Milano nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include "vector_sum.cuh"

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

//////////////////////////////
//////////////////////////////

__global__ void gpu_vector_sum_0(float *x, float *res_tmp, int N) {
    extern __shared__ float shared_data[];
    // each thread loads one element from global to shared mem (warning: no boundary checks!)
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[tid] = i < N ? x[i] : 0;
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) res_tmp[blockIdx.x] = shared_data[0];
}

__global__ void gpu_vector_sum_1(float *x, float *res_tmp, int N) {
    extern __shared__ float shared_data[];
    // each thread loads one element from global to shared mem (warning: no boundary checks!)
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[tid] = i < N ? x[i] : 0;
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            shared_data[index] += shared_data[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) res_tmp[blockIdx.x] = shared_data[0];
}


// Used to sum the values in a warp;
#define WARP_SIZE 32
__inline__ __device__ float warp_reduce(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

// Note: atomicAdd on float is present only in recent GPU architectures.
// If you don't have it, change the benchmark to use floats;
__global__ void gpu_vector_sum_2(float *x, float *res, int N) {
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i];
    }
    sum = warp_reduce(sum);                    // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (WARP_SIZE - 1)) == 0)  // Same as (threadIdx.x % WARP_SIZE) == 0 but faster
        atomicAdd(res, sum);                   // The first thread in the warp updates the output;
}

//////////////////////////////
//////////////////////////////

// Allocate data on the CPU and GPU;
void VectorSum::alloc() {
    // Compute the number of blocks for implementations where the value is a function of the input size;
    B = (N + block_size - 1) / block_size;
    // Allocate CPU data;
    x = (float*) malloc(sizeof(float) * N);
    // hipHostMalloc(&x, sizeof(float) * N); // You can use hipHostMalloc to get faster transfer speed. Don't use it too much though, it slows down CPU memory;
    res_tmp = (float*) malloc(sizeof(float) * B);
    // Allocate GPU data;
    err = hipMalloc(&x_d, sizeof(float) * N);
    // The GPU output buffer has size equal to the number of blocks, 
    // as we aggregate partial sums on the CPU;
    err = hipMalloc(&res_tmp_d, sizeof(float) * B);
}

// Initialize data;
void VectorSum::init() {
    // Just put some values into the array;
    for (int i = 0; i < N; i++) {
        x[i] = float(1) / (i + 1);
    }
}

// Reset the state of the computation after every iteration.
// Reset the result, and transfer data to the GPU if necessary;
void VectorSum::reset() {
    // Reset the result;
    res = 0.0;
    // Transfer data to the GPU;
    hipMemcpy(x_d, x, sizeof(float) * N, hipMemcpyHostToDevice);
    // Reset temporary result vector (required only by some implementations);
    hipMemset(res_tmp_d, 0, sizeof(float) * B);
}

void VectorSum::vector_sum_0(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation (and set the size of shared memory!);
    gpu_vector_sum_0<<<B, block_size, sizeof(float) * block_size>>>(x_d, res_tmp_d, N);
    int second_block_size = (B + block_size - 1) / block_size; // Do it again, to further reduce the amount of data to move back to CPU;
    if (second_block_size > 0) { // If the input data is small enough, this second step is not necessary;
        gpu_vector_sum_0<<<second_block_size, block_size, sizeof(float) * block_size>>>(res_tmp_d, res_tmp_d, B);
    } else {
        second_block_size = B;
    }

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << float(exec_time) / 1000 << " ms, " << (sizeof(float) * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(res_tmp, res_tmp_d, sizeof(float) * second_block_size, hipMemcpyDeviceToHost);
    // Sum the partial results using the CPU;
    for (int i = 0; i < second_block_size; i++) {
        res += res_tmp[i];
    }
}

// Second implementation, without warp divergence;
void VectorSum::vector_sum_1(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation (and set the size of shared memory!);
    gpu_vector_sum_1<<<B, block_size, sizeof(float) * block_size>>>(x_d, res_tmp_d, N);
    int second_block_size = (B + block_size - 1) / block_size; // Do it again, to further reduce the amount of data to move back to CPU;
    if (second_block_size > 0) { // If the input data is small enough, this second step is not necessary;
        gpu_vector_sum_1<<<second_block_size, block_size, sizeof(float) * block_size>>>(res_tmp_d, res_tmp_d, B);
    } else {
        second_block_size = B;
    }

    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << float(exec_time) / 1000 << " ms, " << (sizeof(float) * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(res_tmp, res_tmp_d, sizeof(float) * second_block_size, hipMemcpyDeviceToHost);
    // Sum the partial results using the CPU;
    for (int i = 0; i < second_block_size; i++) {
        res += res_tmp[i];
    }
}

// Third implementation, with grid-stride and shuffle.
// Note: we still use res_tmp_d as output vector,
//   altough this time we use just the first value. 
//   I could have allocated an array of size 1, instead;
void VectorSum::vector_sum_2(int iter) {
    auto start_tmp = clock_type::now();
    // Call the GPU computation;
    gpu_vector_sum_2<<<num_blocks, block_size>>>(x_d, res_tmp_d, N);
    
    // Print performance of GPU, not accounting for transfer time;
    if (debug) {
        // Synchronize computation by hand to measure GPU exec. time;
        hipDeviceSynchronize();
        auto end_tmp = clock_type::now();
        auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        std::cout << "  pure GPU execution(" << iter << ")=" << float(exec_time) / 1000 << " ms, " << (sizeof(float) * N / (exec_time * 1e3)) << " GB/s" << std::endl;
    }

    // Copy the partial result from the GPU to the CPU;
    hipMemcpy(&res, res_tmp_d, sizeof(float), hipMemcpyDeviceToHost);
}

void VectorSum::execute(int iter) {
    switch (implementation)
    {
    case 0:
        vector_sum_0(iter);
        break;
    case 1:
        vector_sum_1(iter);
        break;
    case 2:
        vector_sum_2(iter);
        break;
    default:
        break;
    }
}

void VectorSum::cpu_validation(int iter) {
    auto start_tmp = clock_type::now();
    cpu_result = 0.0;
    for (int i = 0; i < N; i++) {
        cpu_result += x[i];
    }
    auto end_tmp = clock_type::now();
    auto exec_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
    std::cout << "exec time CPU=" << float(exec_time) / 1000 << " ms" << std::endl;
    if (std::abs(res - cpu_result) > 1e-4) std::cout << "result error! GPU=" << res << ", CPU=" << cpu_result << std::endl; 
}

std::string VectorSum::print_result(bool short_form) {
    return std::to_string(res);
}

void VectorSum::clean() {
    free(x);
    free(res_tmp);
    hipFree(x_d);
    hipFree(res_tmp_d);
}
